#include "hip/hip_runtime.h"
/*
 * image_kernels.cu
 *
 * Created on: October 10, 2022
 * (Split from kernels.cu)
 */

#ifndef IMAGE_KERNELS_CU_
#define IMAGE_KERNELS_CU_

#include "common.h"
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <float.h>

namespace kbmod {

/*
 * Device kernel that convolves the provided image with the psf
 */
__global__ void convolvePSF(int width, int height, float *sourceImage,
                            float *resultImage, float *psf, int psfRad, int psfDim,
                            float psfSum, float maskFlag) {
    // Find bounds of convolution area
    const int x = blockIdx.x*CONV_THREAD_DIM+threadIdx.x;
    const int y = blockIdx.y*CONV_THREAD_DIM+threadIdx.y;
    if (x < 0 || x > width-1 || y < 0 || y > height-1) return;

    // Read kernel
    float sum = 0.0;
    float psfPortion = 0.0;
    float center = sourceImage[y*width+x];
    if (center != NO_DATA) {
        for (int j = -psfRad; j <= psfRad; j++) {
            // #pragma unroll
            for (int i = -psfRad; i <= psfRad; i++) {
                if ((x + i >= 0) && (x + i < width) &&
                    (y + j >= 0) && (y + j < height)) {
                    float currentPixel = sourceImage[(y + j) * width + (x + i)];
                    if (currentPixel != NO_DATA) {
                        float currentPSF = psf[(j + psfRad) * psfDim + ( i + psfRad)];
                        psfPortion += currentPSF;
                        sum += currentPixel * currentPSF;
                    }
                }
            }
        }

        resultImage[y*width+x] = (sum * psfSum) / psfPortion;
    } else {
        // Leave masked pixel alone (these could be replaced here with zero)
        resultImage[y*width+x] = NO_DATA; // 0.0
    }
}

extern "C" void deviceConvolve(float *sourceImg, float *resultImg,
                               int width, int height, float *psfKernel,
                               int psfSize, int psfDim, int psfRadius,
                               float psfSum) {
    // Pointers to device memory
    float *deviceKernel;
    float *deviceSourceImg;
    float *deviceResultImg;

    long pixelsPerImage = width*height;
    dim3 blocks(width / CONV_THREAD_DIM + 1, height / CONV_THREAD_DIM + 1);
    dim3 threads(CONV_THREAD_DIM, CONV_THREAD_DIM);

    // Allocate Device memory
    checkCudaErrors(hipMalloc((void **)&deviceKernel, sizeof(float) * psfSize));
    checkCudaErrors(hipMalloc((void **)&deviceSourceImg, sizeof(float) * pixelsPerImage));
    checkCudaErrors(hipMalloc((void **)&deviceResultImg, sizeof(float) * pixelsPerImage));

    checkCudaErrors(hipMemcpy(deviceKernel, psfKernel,
        sizeof(float)*psfSize, hipMemcpyHostToDevice));

    checkCudaErrors(hipMemcpy(deviceSourceImg, sourceImg,
        sizeof(float)*pixelsPerImage, hipMemcpyHostToDevice));

    convolvePSF<<<blocks, threads>>> (width, height, deviceSourceImg,
        deviceResultImg, deviceKernel, psfRadius, psfDim, psfSum, NO_DATA);

    checkCudaErrors(hipMemcpy(resultImg, deviceResultImg,
        sizeof(float)*pixelsPerImage, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(deviceKernel));
    checkCudaErrors(hipFree(deviceSourceImg));
    checkCudaErrors(hipFree(deviceResultImg));
}

// Reads a single pixel from an image buffer
__device__ float readPixel(float* img, int x, int y, int width, int height) {
    return (x < width && y < height) ? img[y * width + x] : NO_DATA;
}

__device__ float maxMasked(float pixel, float previousMax) {
    return pixel == NO_DATA ? previousMax : max(pixel, previousMax);
}

__device__ float minMasked(float pixel, float previousMin) {
    return pixel == NO_DATA ? previousMin : min(pixel, previousMin);
}

/*
 * Reduces the resolution of an image to 1/4 using max pooling
 */
__global__ void pool(int sourceWidth, int sourceHeight, float *source,
                     int destWidth, int destHeight, float *dest, short mode)
{
    const int x = blockIdx.x * POOL_THREAD_DIM + threadIdx.x;
    const int y = blockIdx.y * POOL_THREAD_DIM + threadIdx.y;
    if (x >= destWidth || y >= destHeight)
        return;

    float mp;
    float pixel;
    if (mode == POOL_MAX) {
        mp = -FLT_MAX;
        pixel = readPixel(source, 2 * x, 2 * y, sourceWidth, sourceHeight);
        mp = maxMasked(pixel, mp);
        pixel = readPixel(source, 2 * x + 1, 2 * y, sourceWidth, sourceHeight);
        mp = maxMasked(pixel, mp);
        pixel = readPixel(source, 2 * x, 2 * y + 1, sourceWidth, sourceHeight);
        mp = maxMasked(pixel, mp);
        pixel = readPixel(source, 2 * x + 1, 2 * y + 1, sourceWidth, sourceHeight);
        mp = maxMasked(pixel, mp);
        if (mp == -FLT_MAX) mp = NO_DATA;
    } else {
        mp = FLT_MAX;
        pixel = readPixel(source, 2 * x, 2 * y, sourceWidth, sourceHeight);
        mp = minMasked(pixel, mp);
        pixel = readPixel(source, 2 * x + 1, 2 * y, sourceWidth, sourceHeight);
        mp = minMasked(pixel, mp);
        pixel = readPixel(source, 2 * x, 2 * y + 1, sourceWidth, sourceHeight);
        mp = minMasked(pixel, mp);
        pixel = readPixel(source, 2 * x + 1, 2 * y + 1, sourceWidth, sourceHeight);
        mp = minMasked(pixel, mp);
        if (mp == FLT_MAX)
            mp = NO_DATA;
    }

    dest[y * destWidth + x] = mp;
}

extern "C" void devicePool(int sourceWidth, int sourceHeight, float *source,
                           int destWidth, int destHeight, float *dest,
                           short mode) {
    // Pointers to device memory
    float *deviceSourceImg;
    float *deviceResultImg;

    dim3 blocks(destWidth / POOL_THREAD_DIM + 1, destHeight / POOL_THREAD_DIM + 1);
    dim3 threads(POOL_THREAD_DIM, POOL_THREAD_DIM);

    int srcPixCount = sourceWidth * sourceHeight;
    int destPixCount = destWidth * destHeight;

    // Allocate Device memory
    checkCudaErrors(hipMalloc((void **)&deviceSourceImg,
                               sizeof(float) * srcPixCount));
    checkCudaErrors(hipMalloc((void **)&deviceResultImg,
                               sizeof(float) * destPixCount));

    checkCudaErrors(hipMemcpy(deviceSourceImg, source,
        sizeof(float)*srcPixCount, hipMemcpyHostToDevice));

    pool<<<blocks, threads>>> (sourceWidth, sourceHeight, deviceSourceImg,
            destWidth, destHeight, deviceResultImg, mode);

    checkCudaErrors(hipMemcpy(dest, deviceResultImg,
        sizeof(float)*destPixCount, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(deviceSourceImg));
    checkCudaErrors(hipFree(deviceResultImg));
}

/*
 * Uses pooling to extend min/max regions without reducing the resolution
 * of the image.
 */
__global__ void pool_in_place(int width, int height, float *source, float *dest,
                              int radius, short mode) {
    const int x = blockIdx.x * POOL_THREAD_DIM + threadIdx.x;
    const int y = blockIdx.y * POOL_THREAD_DIM + threadIdx.y;
    if (x >= width || y >= height)
        return;

    float mp = NO_DATA;
    float pixel;

    // Compute the bounds over which to pool.
    int xs = max(x - radius, 0);
    int xe = min(x + radius, width - 1);
    int ys = max(y - radius, 0);
    int ye = min(y + radius, height - 1);

    if (mode == POOL_MAX) {
        mp = -FLT_MAX;
        for (int xi = xs; xi <= xe; ++xi) {
            for (int yi = ys; yi <= ye; ++yi) {
                pixel = source[yi * width + xi];
                mp = (pixel == NO_DATA) ? mp : max(pixel, mp);
            }
        }
        if (mp == -FLT_MAX)
            mp = NO_DATA;
    } else {
        mp = FLT_MAX;
        for (int xi = xs; xi <= xe; ++xi) {
            for (int yi = ys; yi <= ye; ++yi) {
                pixel = source[yi * width + xi];
                mp = (pixel == NO_DATA) ? mp : min(pixel, mp);
            }
        }
        if (mp == FLT_MAX)
            mp = NO_DATA;
    }

    dest[y * width + x] = mp;
}

extern "C" void devicePoolInPlace(int width, int height, float *source, float *dest,
                                  int radius, short mode)
{
    // Pointers to device memory
    float *deviceSourceImg;
    float *deviceResultImg;

    int pixCount = width * height;
    dim3 blocks(width / POOL_THREAD_DIM + 1, height / POOL_THREAD_DIM + 1);
    dim3 threads(POOL_THREAD_DIM, POOL_THREAD_DIM);

    // Allocate Device memory
    checkCudaErrors(hipMalloc((void **)&deviceSourceImg,
                               sizeof(float) * pixCount));
    checkCudaErrors(hipMalloc((void **)&deviceResultImg,
                               sizeof(float) * pixCount));

    // Copy the source image into GPU memory.
    checkCudaErrors(hipMemcpy(deviceSourceImg, source,
                               sizeof(float)*pixCount,
                               hipMemcpyHostToDevice));

    pool_in_place<<<blocks, threads>>> (width, height, deviceSourceImg,
                                        deviceResultImg, radius, mode);

    // Copy the final image from GPU memory to dest.
    checkCudaErrors(hipMemcpy(dest, deviceResultImg,
                               sizeof(float)*pixCount,
                               hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(deviceSourceImg));
    checkCudaErrors(hipFree(deviceResultImg));
}

__global__ void grow_mask(int width, int height, float *source, 
                          float *dest, int steps) {
    const int x = blockIdx.x * POOL_THREAD_DIM + threadIdx.x;
    const int y = blockIdx.y * POOL_THREAD_DIM + threadIdx.y;
    if (x >= width || y >= height)
        return;

    // Get the original pixel value.
    float pixel_val = source[y * width + x];

    // Check each pixel within steps distance.
    int ys = max(0, y - steps);
    int ye = min(height - 1, y + steps);
    for (int yi = ys; yi <= ye; ++yi) {
        int steps_left = steps - abs(y - yi);
        int xs = max(0, x - steps_left);
        int xe = min(width - 1, x + steps_left);
        
        for (int xi = xs; xi <= xe; ++xi){
            if (source[yi * width + xi] == NO_DATA)
                pixel_val = NO_DATA;
        }
    }

    dest[y * width + x] = pixel_val;
}

extern "C" void deviceGrowMask(int width, int height, float *source, 
                               float *dest, int steps) {
    // Pointers to device memory
    float *deviceSourceImg;
    float *deviceResultImg;

    int pixCount = width * height;
    dim3 blocks(width / POOL_THREAD_DIM + 1, height / POOL_THREAD_DIM + 1);
    dim3 threads(POOL_THREAD_DIM, POOL_THREAD_DIM);

    // Allocate Device memory
    checkCudaErrors(hipMalloc((void **)&deviceSourceImg,
                               sizeof(float) * pixCount));
    checkCudaErrors(hipMalloc((void **)&deviceResultImg,
                               sizeof(float) * pixCount));

    // Copy the source image into GPU memory.
    checkCudaErrors(hipMemcpy(deviceSourceImg, source,
                               sizeof(float)*pixCount,
                               hipMemcpyHostToDevice));

    grow_mask<<<blocks, threads>>> (width, height, deviceSourceImg,
                                    deviceResultImg, steps);

    // Copy the final image from GPU memory to dest.
    checkCudaErrors(hipMemcpy(dest, deviceResultImg,
                               sizeof(float)*pixCount,
                               hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(deviceSourceImg));
    checkCudaErrors(hipFree(deviceResultImg));
}


} /* namespace kbmod */

#endif /* IMAGE_KERNELS_CU_ */
