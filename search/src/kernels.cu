#include "hip/hip_runtime.h"
/*
 * kernels.cu
 *
 *  Created on: Jun 20, 2017
 *      Author: kbmod-usr
 */

#ifndef KERNELS_CU_
#define KERNELS_CU_
#define GPU_LC_FILTER 1
#define MAX_NUM_IMAGES 140

#include "common.h"
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <float.h>
#include "filtering_kernels.cu"

namespace kbmod {


/*
 * Device kernel that convolves the provided image with the psf
 */
__global__ void convolvePSF(int width, int height,
    float *sourceImage, float *resultImage, float *psf,
    int psfRad, int psfDim, float psfSum, float maskFlag)
{
    // Find bounds of convolution area
    const int x = blockIdx.x*CONV_THREAD_DIM+threadIdx.x;
    const int y = blockIdx.y*CONV_THREAD_DIM+threadIdx.y;
    if (x < 0 || x > width-1 || y < 0 || y > height-1) return;

    // Read kernel
    float sum = 0.0;
    float psfPortion = 0.0;
    float center = sourceImage[y*width+x];
    if (center != NO_DATA) {
        for (int j = -psfRad; j <= psfRad; j++)
        {
            // #pragma unroll
            for (int i = -psfRad; i <= psfRad; i++)
            {
                if ((x + i >= 0) && (x + i < width) &&
                    (y + j >= 0) && (y + j < height))
                {
                    float currentPixel = sourceImage[(y+j)*width+(x+i)];
                    if (currentPixel != NO_DATA)
                    {
                        float currentPSF = psf[(j+psfRad)*psfDim+(i+psfRad)];
                        psfPortion += currentPSF;
                        sum += currentPixel * currentPSF;
                    }
                }
            }
        }

        resultImage[y*width+x] = (sum*psfSum)/psfPortion;
    } else {
        // Leave masked pixel alone (these could be replaced here with zero)
        resultImage[y*width+x] = NO_DATA; // 0.0
    }
}

extern "C" void deviceConvolve(float *sourceImg, float *resultImg,
    int width, int height, float *psfKernel,
    int psfSize, int psfDim, int psfRadius, float psfSum)
{
    // Pointers to device memory //
    float *deviceKernel;
    float *deviceSourceImg;
    float *deviceResultImg;

    long pixelsPerImage = width*height;
    dim3 blocks(width/CONV_THREAD_DIM+1,height/CONV_THREAD_DIM+1);
    dim3 threads(CONV_THREAD_DIM,CONV_THREAD_DIM);

    // Allocate Device memory
    checkCudaErrors(hipMalloc((void **)&deviceKernel, sizeof(float)*psfSize));
    checkCudaErrors(hipMalloc((void **)&deviceSourceImg, sizeof(float)*pixelsPerImage));
    checkCudaErrors(hipMalloc((void **)&deviceResultImg, sizeof(float)*pixelsPerImage));

    checkCudaErrors(hipMemcpy(deviceKernel, psfKernel,
        sizeof(float)*psfSize, hipMemcpyHostToDevice));

    checkCudaErrors(hipMemcpy(deviceSourceImg, sourceImg,
        sizeof(float)*pixelsPerImage, hipMemcpyHostToDevice));

    convolvePSF<<<blocks, threads>>> (width, height, deviceSourceImg,
        deviceResultImg, deviceKernel, psfRadius, psfDim, psfSum, NO_DATA);

    checkCudaErrors(hipMemcpy(resultImg, deviceResultImg,
        sizeof(float)*pixelsPerImage, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(deviceKernel));
    checkCudaErrors(hipFree(deviceSourceImg));
    checkCudaErrors(hipFree(deviceResultImg));
}

// Reads a single pixel from an image buffer
__device__ float readPixel(float* img, int x, int y, int width, int height)
{
    return (x<width && y<height) ? img[y*width+x] : NO_DATA;
}

__device__ float maxMasked(float pixel, float previousMax)
{
    return pixel == NO_DATA ? previousMax : max(pixel, previousMax);
}

__device__ float minMasked(float pixel, float previousMin)
{
    return pixel == NO_DATA ? previousMin : min(pixel, previousMin);
}

/*
 * Reduces the resolution of an image to 1/4 using max pooling
 */
__global__ void pool(int sourceWidth, int sourceHeight, float *source,
    int destWidth, int destHeight, float *dest, short mode)
{
    const int x = blockIdx.x*POOL_THREAD_DIM+threadIdx.x;
    const int y = blockIdx.y*POOL_THREAD_DIM+threadIdx.y;
    if (x>=destWidth || y>=destHeight) return;
    float mp;
    float pixel;
    if (mode == POOL_MAX) {
        mp = -FLT_MAX;
        pixel = readPixel(source, 2*x,   2*y,   sourceWidth, sourceHeight);
        mp = maxMasked(pixel, mp);
        pixel = readPixel(source, 2*x+1, 2*y,   sourceWidth, sourceHeight);
        mp = maxMasked(pixel, mp);
        pixel = readPixel(source, 2*x,   2*y+1, sourceWidth, sourceHeight);
        mp = maxMasked(pixel, mp);
        pixel = readPixel(source, 2*x+1, 2*y+1, sourceWidth, sourceHeight);
        mp = maxMasked(pixel, mp);
        if (mp == -FLT_MAX) mp = NO_DATA;
    } else {
        mp = FLT_MAX;
        pixel = readPixel(source, 2*x,   2*y,   sourceWidth, sourceHeight);
        mp = minMasked(pixel, mp);
        pixel = readPixel(source, 2*x+1, 2*y,   sourceWidth, sourceHeight);
        mp = minMasked(pixel, mp);
        pixel = readPixel(source, 2*x,   2*y+1, sourceWidth, sourceHeight);
        mp = minMasked(pixel, mp);
        pixel = readPixel(source, 2*x+1, 2*y+1, sourceWidth, sourceHeight);
        mp = minMasked(pixel, mp);
        if (mp == FLT_MAX) mp = NO_DATA;
    }

    dest[y*destWidth+x] = mp;
}

extern "C" void devicePool(int sourceWidth, int sourceHeight, float *source,
    int destWidth, int destHeight, float *dest, short mode)
{
    // Pointers to device memory //
    float *deviceSourceImg;
    float *deviceResultImg;

    dim3 blocks(destWidth/POOL_THREAD_DIM+1,destHeight/POOL_THREAD_DIM+1);
    dim3 threads(POOL_THREAD_DIM,POOL_THREAD_DIM);

    int srcPixCount = sourceWidth*sourceHeight;
    int destPixCount = destWidth*destHeight;

    // Allocate Device memory
    checkCudaErrors(hipMalloc((void **)&deviceSourceImg, sizeof(float)*srcPixCount));
    checkCudaErrors(hipMalloc((void **)&deviceResultImg, sizeof(float)*destPixCount));

    checkCudaErrors(hipMemcpy(deviceSourceImg, source,
        sizeof(float)*srcPixCount, hipMemcpyHostToDevice));

    pool<<<blocks, threads>>> (sourceWidth, sourceHeight, deviceSourceImg,
            destWidth, destHeight, deviceResultImg, mode);

    checkCudaErrors(hipMemcpy(dest, deviceResultImg,
        sizeof(float)*destPixCount, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(deviceSourceImg));
    checkCudaErrors(hipFree(deviceResultImg));
}

/*
 * Searches through images (represented as a flat array of floats) looking for most likely
 * trajectories in the given list. Outputs a results image of best trajectories. Returns a
 * fixed number of results per pixel specified by RESULTS_PER_PIXEL
 * filters results using a sigmaG-based filter and a central-moment filter.
 */
__global__ void searchFilterImages(int trajectoryCount, int width, int height,
        int imageCount, int minObservations, float *psiPhiImages,
        trajectory *trajectories, trajectory *results, float *imgTimes,
        bool doFilter, float sGL0, float sGL1, float sigmaGCoeff, float minLH,
        bool useCorr, baryCorrection *baryCorrs)
{
    // Get origin pixel for the trajectories.
    const unsigned short x = blockIdx.x*THREAD_DIM_X+threadIdx.x;
    const unsigned short y = blockIdx.y*THREAD_DIM_Y+threadIdx.y;

    // Data structures used for filtering.
    float lcArray[MAX_NUM_IMAGES];
    float psiArray[MAX_NUM_IMAGES];
    float phiArray[MAX_NUM_IMAGES];
    int idxArray[MAX_NUM_IMAGES];

    // Create an initial set of best results with likelihood -1.0.
    trajectory best[RESULTS_PER_PIXEL];
    for (int r=0; r < RESULTS_PER_PIXEL; ++r)
    {
        best[r].lh = -1.0;
    }
    
    // Give up on any trajectories starting outside the image
    if (x >= width || y >= height)
    {
        return;
    }

    const unsigned int pixelsPerImage = width*height;

    // Use a shared array of times that is cached as opposed
    // to constantly reading from global memory.
    __shared__ float sImgTimes[512];
    int idx = threadIdx.x+threadIdx.y*THREAD_DIM_X;
    if (idx<imageCount) sImgTimes[idx] = imgTimes[idx];
    __syncthreads();

    // For each trajectory we'd like to search
    for (int t=0; t < trajectoryCount; ++t)
    {
        // Create a trajectory for this search.
        trajectory currentT;
        currentT.x = x;
        currentT.y = y;
        currentT.xVel = trajectories[t].xVel;
        currentT.yVel = trajectories[t].yVel;
        currentT.obsCount = 0;

        float psiSum = 0.0;
        float phiSum = 0.0;

        // Reset everything a default values.
        for (int i = 0; i < imageCount; ++i)
        {
            lcArray[i] = 0;
            psiArray[i] = 0;
            phiArray[i] = 0;
            idxArray[i] = i;
        }
        
        // Loop over each image and sample the appropriate pixel
        int num_seen = 0;
        for (int i = 0; i < imageCount; ++i)
        {
            // Predict the trajectory's position.
            float cTime = sImgTimes[i];
            int currentX = x + int(currentT.xVel*cTime+0.5);
            int currentY = y + int(currentT.yVel*cTime+0.5);

            // If using barycentric correction, apply it
            // This branch is short, and all threads should
            // have same value of baryCorr, so hopefully
            // performance is OK?
            // Must be before out of bounds check
            if (useCorr) {
                baryCorrection bc = baryCorrs[i];
                currentX = int(x + currentT.xVel*cTime + bc.dx + x*bc.dxdx + y*bc.dxdy + 0.5);
                currentY = int(y + currentT.yVel*cTime + bc.dy + x*bc.dydx + y*bc.dydy + 0.5);
            }
                
            // Test if trajectory goes out of image bounds
            // Branching could be avoided here by setting a
            // black image border and clamping coordinates
            if (currentX >= width || currentY >= height
                || currentX < 0 || currentY < 0)
            {
                continue;
            }

            // Get the Psi and Phi pixel values.
            unsigned int pixel_index = (pixelsPerImage*i + currentY*width
                                        + currentX);
            float2 cPsiPhi = reinterpret_cast<float2*>(psiPhiImages)[pixel_index];

            // Only aggregate the sums and fill in the arrays if
            // we are seeing a non-masked point. Otherwise skip it.
            if (cPsiPhi.x == NO_DATA) continue;

            currentT.obsCount++;
            psiSum += cPsiPhi.x;
            phiSum += cPsiPhi.y;
            psiArray[num_seen] = cPsiPhi.x;
            phiArray[num_seen] = cPsiPhi.y;
            if (cPsiPhi.y == 0.0)
            {
                lcArray[num_seen] = 0.0;
            } else {
                lcArray[num_seen] = cPsiPhi.x/cPsiPhi.y;
            }
            num_seen += 1;
        }
        currentT.lh = psiSum/sqrt(phiSum);
        currentT.flux = psiSum/phiSum;

        // If we don't have enough observations or do not meet the
        // minLH threshold (and are doing filtering) just stop now.
        // It's not worth doing the sigmaG filtering or inserting into
        // the results.
        if ((currentT.obsCount < minObservations) ||
            (doFilter && (currentT.lh < minLH)))
        {
            continue;
        }

        // If we are doing on GPU filtering, run the sigmaG filter
        // and recompute the likelihoods.
        if (doFilter)
        {
            int minKeepIndex = 0;
            int maxKeepIndex = num_seen - 1;
            sigmaGFilteredIndicesCU(lcArray, num_seen, sGL0, sGL1, sigmaGCoeff,
                                    2.0, idxArray, &minKeepIndex, &maxKeepIndex);

            // Compute the likelihood and flux of the track based on the filtered
            // observations (ones in [minKeepIndex, maxKeepIndex]).
            float newPsiSum = 0.0;
            float newPhiSum = 0.0;
            for (int i = minKeepIndex; i <= maxKeepIndex; i++)
            {
                int idx = idxArray[i];
                newPsiSum += psiArray[idx];
                newPhiSum += phiArray[idx];
            }

            // Compute the new likelihood and filter if needed.
            currentT.lh = newPsiSum/sqrt(newPhiSum);
            currentT.flux = newPsiSum/newPhiSum;
        }

        // Insert the new trajectory into the sorted list of results.
        trajectory temp;
        for (int r = 0; r < RESULTS_PER_PIXEL; ++r)
        {
            if (currentT.lh > best[r].lh &&
                currentT.obsCount >= minObservations)
            {
                temp = best[r];
                best[r] = currentT;
                currentT = temp;
            }
        }
    }
    
    // Copy the sorted list of best results for this pixel into
    // the correct location within the global results vector.
    const int base_index = (y * width + x) * RESULTS_PER_PIXEL;
    for (int r = 0; r < RESULTS_PER_PIXEL; ++r)
    {
        results[base_index + r] = best[r];
    }
}

extern "C" void
deviceSearchFilter(
        int trajCount, int imageCount, int minObservations, int psiPhiSize,
        int resultsCount, trajectory *trajectoriesToSearch, trajectory *bestTrajects,
        float *imageTimes, float *interleavedPsiPhi, int width, int height,
        bool doFilter, float sigmaGLims[2], float sigmaGCoeff, float minLH,
        bool useCorr, baryCorrection *baryCorrs)
{
    // Allocate Device memory
    trajectory *deviceTests;
    float *deviceImgTimes;
    float *devicePsiPhi;
    trajectory *deviceSearchResults;

    checkCudaErrors(hipMalloc((void **)&deviceTests, sizeof(trajectory)*trajCount));
    checkCudaErrors(hipMalloc((void **)&deviceImgTimes, sizeof(float)*imageCount));
    checkCudaErrors(hipMalloc((void **)&devicePsiPhi,
        sizeof(float)*psiPhiSize));
    checkCudaErrors(hipMalloc((void **)&deviceSearchResults,
        sizeof(trajectory)*resultsCount));

    // Copy trajectories to search
    checkCudaErrors(hipMemcpy(deviceTests, trajectoriesToSearch,
            sizeof(trajectory)*trajCount, hipMemcpyHostToDevice));

    // Copy image times
    checkCudaErrors(hipMemcpy(deviceImgTimes, imageTimes,
            sizeof(float)*imageCount, hipMemcpyHostToDevice));

    // Copy interleaved buffer of psi and phi images
    checkCudaErrors(hipMemcpy(devicePsiPhi, interleavedPsiPhi,
        sizeof(float)*psiPhiSize, hipMemcpyHostToDevice));

    // allocate memory for and copy barycentric corrections
    baryCorrection* deviceBaryCorrs;
    if (useCorr) {
        checkCudaErrors(hipMalloc((void **)&deviceBaryCorrs,
            sizeof(baryCorrection)*imageCount));
        checkCudaErrors(hipMemcpy(deviceBaryCorrs, baryCorrs,
            sizeof(baryCorrection)*imageCount, hipMemcpyHostToDevice));
    }

    dim3 blocks(width/THREAD_DIM_X+1,height/THREAD_DIM_Y+1);
    dim3 threads(THREAD_DIM_X,THREAD_DIM_Y);


    // Launch Search
    searchFilterImages<<<blocks, threads>>> (trajCount, width,
        height, imageCount, minObservations, devicePsiPhi,
        deviceTests, deviceSearchResults, deviceImgTimes, 
        doFilter, sigmaGLims[0], sigmaGLims[1], sigmaGCoeff, minLH, 
        useCorr, deviceBaryCorrs);

    // Read back results
    checkCudaErrors(hipMemcpy(bestTrajects, deviceSearchResults,
                sizeof(trajectory)*resultsCount, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(deviceTests));
    checkCudaErrors(hipFree(deviceImgTimes));
    checkCudaErrors(hipFree(deviceSearchResults));
    checkCudaErrors(hipFree(devicePsiPhi));

    if (useCorr){
        checkCudaErrors(hipFree(deviceBaryCorrs));
    }
}

} /* namespace kbmod */

#endif /* KERNELS_CU_ */
